#include "hip/hip_runtime.h"
/******************************************************************************

  usage information and TODO stuff here...

******************************************************************************/


#include <stdio.h>
#include "gpu.h"

/******************************************************************************/
APopulation initializePop(unsigned int numBlocks, unsigned int numThreads){

  APopulation thePop;

  thePop.nBlocks = numBlocks;
  thePop.nThreads = numThreads;
  thePop.N = numBlocks * numThreads;

  hipMalloc( (void**) &thePop.dev_a, thePop.N*sizeof(int));
  hipMalloc( (void**) &thePop.dev_b, thePop.N*sizeof(int));
  hipMalloc( (void**) &thePop.dev_c, thePop.N*sizeof(int));

  //----- placeholder for initializing memory with values
  int a[thePop.N], b[thePop.N];
  for (int i=0; i<thePop.N; i++){
    a[i] = -i;
    b[i] = i*i;
  }
  hipMemcpy(thePop.dev_a, a, thePop.N*sizeof(int), cH2D);
  hipMemcpy(thePop.dev_b, b, thePop.N*sizeof(int), cH2D);
  // ------------------------

  return thePop;
}


/******************************************************************************/
__global__ void add(int *a, int *b, int *c){

  int tid = threadIdx.x + (blockIdx.x * blockDim.x);

  c[tid] = a[tid] + b[tid];

}

/******************************************************************************/
int runIter(APopulation *thePop){

  add <<< thePop->nBlocks, thePop->nThreads >>>
                                (thePop->dev_a, thePop->dev_b, thePop->dev_c);

  // -- crud...
  int a[thePop->N], b[thePop->N], c[thePop->N];
  hipMemcpy(&a, thePop->dev_a, thePop->N * sizeof(int), cD2H);
  hipMemcpy(&b, thePop->dev_b, thePop->N * sizeof(int), cD2H);
  hipMemcpy(&c, thePop->dev_c, thePop->N * sizeof(int), cD2H);

  for(int i = 0; i< thePop->N; i++){
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
  // ----

  return 0;
}


/******************************************************************************/
void freeGPU(APopulation *thePop)
{
  hipFree(thePop->dev_a);
  hipFree(thePop->dev_b);
  hipFree(thePop->dev_c);
}

/******************************************************************************/
