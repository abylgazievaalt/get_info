#include <stdio.h>
#include <hip/hip_runtime.h>

# define cD2H hipMemcpyDeviceToHost
# define cH2D hipMemcpyHostToDevice

# define N 65000
# define NumThreads 1

/*****************************************************/
__global__ void add(int *a, int *b, int *c){

  int tid = blockIdx.x;

  c[tid] = a[tid] + b[tid];

}


/*****************************************************/
int main(){

  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  for (int i=0; i<N; i++){
    a[i] = -i;
    b[i] = i*i;
  }

  hipMalloc( (void**)&dev_a, N*sizeof(int));
  hipMalloc( (void**)&dev_b, N*sizeof(int));
  hipMalloc( (void**)&dev_c, N*sizeof(int));

  hipMemcpy(dev_a, a, N*sizeof(int), cH2D);
  hipMemcpy(dev_b, b, N*sizeof(int), cH2D);

  add <<< N, NumThreads >>> (dev_a, dev_b, dev_c);

  hipMemcpy(&c, dev_c, N*sizeof(int), cD2H);

  for(int i = 0; i< N; i++){
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
